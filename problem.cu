#include "hip/hip_runtime.h"
#include "header.h"
#include <hip/hip_runtime.h>

using namespace std;

#define BLOCK_DIM 16
#define dt_h      1e-2
#define dt_m      1e-6

__global__ void kernel_SetIC(DAT *Txx, DAT *Tyy, DAT *Txy, DAT *Vx, DAT *Vy, DAT *Ux, DAT *Uy, DAT *rsd_m_x, DAT *rsd_m_y, DAT *Pw, DAT *Sw, DAT *qx, DAT *qy, DAT *Krx, DAT *Kry, DAT *rsd_h,
                             const int nx, const int ny, const DAT Lx, const DAT Ly);
__global__ void kernel_Compute_Sw(DAT *Pw, DAT *Sw, const int nx, const int ny,
                             const DAT rhow, const DAT g,
                             const DAT vg_a, const DAT vg_m, const DAT vg_n);
__global__ void kernel_Compute_Q(DAT *qx, DAT *qy, DAT *Pw, DAT *Krx, DAT *Kry,
                                 const int nx, const int ny, const DAT dx, const DAT dy,
                                 const DAT K,  const DAT rhow, const DAT muw, const DAT g);
__global__ void kernel_Compute_Kr(DAT *qx, DAT *qy, DAT *Pw, DAT *Sw, DAT *Krx, DAT *Kry,
                                 const int nx, const int ny, const DAT vg_m);

__global__ void kernel_Update_Pw(DAT *rsd, DAT *Pw, DAT *Sw, DAT *Pw_old, DAT *Sw_old,
                                 DAT *qx, DAT *qy, const int nx, const int ny,
                                 const DAT dx, const DAT dy, const DAT dt,
                                 const DAT phi, const DAT rhow, const DAT sstor);

__global__ void kernel_Update_V(DAT *Vx, DAT *Vy, DAT *Txx, DAT *Tyy, DAT *Txy, DAT *Pw, DAT *Sw,
                                const int nx, const int ny, const DAT dx, const DAT dy,
                                const DAT rho_s, const DAT g);

__global__ void kernel_Update_U(DAT *Ux, DAT *Uy, DAT *Vx, DAT *Vy, const int nx, const int ny);
__global__ void kernel_Update_Stress(DAT *Txx, DAT *Tyy, DAT *Txy, DAT *Vx, DAT *Vy, DAT *Pw, DAT *Sw, DAT *Sw_old, DAT *rsd_m_x, DAT *rsd_m_y, const int nx, const int ny,
                                 const DAT dx,  const DAT dy, const DAT dt,
                                 const DAT rho_s, const DAT g, const DAT mu, const DAT lam);

void Problem::SetIC_GPU()
{
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid((nx+dimBlock.x-1)/dimBlock.x, (ny+dimBlock.y-1)/dimBlock.y);
    printf("Launching %dx%d blocks of %dx%d threads\n", (nx+2+dimBlock.x-1)/dimBlock.x,
           (ny+2+dimBlock.y-1)/dimBlock.y, BLOCK_DIM, BLOCK_DIM);
    //hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    kernel_SetIC<<<dimGrid,dimBlock>>>(dev_Txx, dev_Tyy, dev_Txy,
                                       dev_Vx, dev_Vy, dev_Ux, dev_Uy,
                                       dev_rsd_m_x, dev_rsd_m_y,
                                       dev_Pw, dev_Sw, dev_qx, dev_qy, dev_Krx, dev_Kry, dev_rsd_h,
                                       nx, ny, Lx, Ly);
    hipError_t err = hipGetLastError();
    if(err != 0)
        printf("Error %x at SetIC\n", err);
    //kernel_Compute_Sw<<<dimGrid,dimBlock>>>(Pw, Sw, rhow, g, vg_a, vg_m, vg_n);
    //kernel_ComputeKr<<<dimGrid,dimBlock>>>(H, Theta, Krx, Kry, nx, ny, dy);
    //kernel_ComputeFluidFluxes<<<dimGrid,dimBlock>>>(H, qx, qy, Krx, Kry, nx, ny, dx, dy, D);
    Compute_Sw_GPU();
    //Compute_Kr_GPU();
    //Compute_Q_GPU();
}

void Problem::Update_V_GPU()
{
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid((nx+1+dimBlock.x-1)/dimBlock.x, (ny+1+dimBlock.y-1)/dimBlock.y);
    kernel_Update_V<<<dimGrid,dimBlock>>>(dev_Vx, dev_Vy, dev_Txx, dev_Tyy, dev_Txy,
                                          dev_Pw, dev_Sw,
                                          nx, ny, dx, dy, rho_s, g);
    hipError_t err = hipGetLastError();
    if(err != 0)
        printf("Error %x at V\n", err);
}

void Problem::Update_U_GPU()
{
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid((nx+1+dimBlock.x-1)/dimBlock.x, (ny+1+dimBlock.y-1)/dimBlock.y);
    kernel_Update_U<<<dimGrid,dimBlock>>>(dev_Ux, dev_Uy, dev_Vx, dev_Vy, nx, ny);
    hipError_t err = hipGetLastError();
    if(err != 0)
        printf("Error %x at U\n", err);
}

void Problem::Update_Stress_GPU()
{
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid((nx+dimBlock.x-1)/dimBlock.x, (ny+dimBlock.y-1)/dimBlock.y);
    kernel_Update_Stress<<<dimGrid,dimBlock>>>(dev_Txx, dev_Tyy, dev_Txy,
                                               dev_Vx, dev_Vy, dev_Pw, dev_Sw, dev_Sw_old,
                                               dev_rsd_m_x, dev_rsd_m_y,
                                               nx, ny, dx, dy, dt, rho_s, g, mu, lam);
    hipError_t err = hipGetLastError();
    if(err != 0)
        printf("Error %x at Stress\n", err);
}

void Problem::Compute_Sw_GPU()
{
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid((nx+dimBlock.x-1)/dimBlock.x, (ny+dimBlock.y-1)/dimBlock.y);
    kernel_Compute_Sw<<<dimGrid,dimBlock>>>(dev_Pw, dev_Sw, nx, ny, rhow, g, vg_a, vg_m, vg_n);
    hipError_t err = hipGetLastError();
    if(err != 0)
        printf("Error %x\n", err);
}

void Problem::Compute_Q_GPU()
{
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid((nx+1+dimBlock.x-1)/dimBlock.x, (ny+1+dimBlock.y-1)/dimBlock.y);
    kernel_Compute_Q<<<dimGrid,dimBlock>>>(dev_qx, dev_qy, dev_Pw, dev_Krx, dev_Kry,
                                           nx, ny, dx, dy, K, rhow, muw, g);
    hipError_t err = hipGetLastError();
    if(err != 0)
        printf("Error %x at Q\n", err);
}

void Problem::Compute_Kr_GPU()
{
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid((nx+1+dimBlock.x-1)/dimBlock.x, (ny+1+dimBlock.y-1)/dimBlock.y);
    kernel_Compute_Kr<<<dimGrid,dimBlock>>>(dev_qx, dev_qy, dev_Pw, dev_Sw, dev_Krx, dev_Kry,
                                            nx, ny, vg_m);
    hipError_t err = hipGetLastError();
    if(err != 0)
        printf("Error %x at Kr\n", err);
}

void Problem::Update_Pw_GPU()
{
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid((nx+dimBlock.x-1)/dimBlock.x, (ny+dimBlock.y-1)/dimBlock.y);
    kernel_Update_Pw<<<dimGrid,dimBlock>>>(dev_rsd_h, dev_Pw, dev_Sw, dev_Pw_old, dev_Sw_old,
                                           dev_qx, dev_qy, nx, ny, dx, dy, dt,
                                           phi, rhow, sstor);
    hipError_t err = hipGetLastError();
    if(err != 0)
        printf("Error %x at Pw\n", err);
}

void Problem::M_Substep_GPU()
{
    printf("Mechanics\n");
    fflush(stdout);
    for(int nit = 1; nit < 50000; nit++){
        Update_V_GPU();
        Update_U_GPU();
        Update_Stress_GPU();
        if(nit%10000 == 0 || nit == 1){
            hipMemcpy(rsd_m_x, dev_rsd_m_x, sizeof(DAT) * nx*ny, hipMemcpyDeviceToHost);
            hipMemcpy(rsd_m_y, dev_rsd_m_y, sizeof(DAT) * nx*ny, hipMemcpyDeviceToHost);
            DAT err_m_x = 0.;
            DAT err_m_y = 0.;
            for(int i = 0; i < nx*ny; i++){
                if(fabs(rsd_m_x[i]) > err_m_x)
                    err_m_x = fabs(rsd_m_x[i]);
                if(fabs(rsd_m_y[i]) > err_m_y)
                    err_m_y = fabs(rsd_m_y[i]);

                if(isinf(rsd_m_x[i]) || isnan(rsd_m_x[i]) || isinf(rsd_m_y[i]) || isnan(rsd_m_y[i])){
                    printf("Bad value, iter %d", nit);
                    exit(0);
                }
            }
            printf("iter %d: r_m_x = %e, r_m_y = %e\n", nit, err_m_x, err_m_y);
            fflush(stdout);
            if(err_m_x < eps_a_m && err_m_y < eps_a_m){
                printf("Mechanics converged in %d it.: r_m_x = %e, r_m_y = %e\n", nit, err_m_x, err_m_y);
                break;
            }
        }
    }

}

void Problem::H_Substep_GPU()
{
    printf("Flow\n");
    hipMemcpy(dev_Pw_old, dev_Pw, sizeof(DAT) * nx*ny, hipMemcpyDeviceToDevice);
    hipMemcpy(dev_Sw_old, dev_Sw, sizeof(DAT) * nx*ny, hipMemcpyDeviceToDevice);
    for(int nit = 1; nit < 100000; nit++){
        Compute_Sw_GPU();
        Compute_Kr_GPU();
        Compute_Q_GPU();
        Update_Pw_GPU();
        if(nit%10000 == 0 || nit == 1){
            hipMemcpy(rsd_h, dev_rsd_h, sizeof(DAT) * nx*ny, hipMemcpyDeviceToHost);
            DAT err = 0;
            for(int i = 0; i < nx*ny; i++){
                if(fabs(rsd_h[i]) > err)
                    err = fabs(rsd_h[i]);
                if(isinf(rsd_h[i]) || isnan(rsd_h[i])){
                    printf("Bad value, iter %d", nit);
                    exit(0);
                }
            }
            printf("iter %d: r_w = %e\n", nit, err);
            fflush(stdout);
            if(err < eps_a_h){
                printf("Flow converged in %d it.: r_w = %e\n", nit, err);
                break;
            }
        }
    }
}

void Problem::SolveOnGPU()
{
    hipMalloc((void**)&dev_Pw,     sizeof(DAT) * nx*ny);
    hipMalloc((void**)&dev_Sw,     sizeof(DAT) * nx*ny);
    hipMalloc((void**)&dev_Pw_old, sizeof(DAT) * nx*ny);
    hipMalloc((void**)&dev_Sw_old, sizeof(DAT) * nx*ny);
    hipMalloc((void**)&dev_qx,     sizeof(DAT) * (nx+1)*ny);
    hipMalloc((void**)&dev_qy,     sizeof(DAT) * nx*(ny+1));
    hipMalloc((void**)&dev_Krx,    sizeof(DAT) * (nx+1)*ny);
    hipMalloc((void**)&dev_Kry,    sizeof(DAT) * nx*(ny+1));
    hipMalloc((void**)&dev_rsd_h,  sizeof(DAT) * nx*ny);

    hipMalloc((void**)&dev_Txx,    sizeof(DAT) * nx*ny);
    hipMalloc((void**)&dev_Tyy,    sizeof(DAT) * nx*ny);
    hipMalloc((void**)&dev_Txy,    sizeof(DAT) * (nx+1)*(ny+1));
    hipMalloc((void**)&dev_Vx,     sizeof(DAT) * (nx+1)*ny);
    hipMalloc((void**)&dev_Vy,     sizeof(DAT) * nx*(ny+1));
    hipMalloc((void**)&dev_Ux,     sizeof(DAT) * (nx+1)*ny);
    hipMalloc((void**)&dev_Uy,     sizeof(DAT) * nx*(ny+1));
    hipMalloc((void**)&dev_rsd_m_x,  sizeof(DAT) * nx*ny);
    hipMalloc((void**)&dev_rsd_m_y,  sizeof(DAT) * nx*ny);

    printf("Allocated on GPU\n");

    // Still needed for VTK saving
    Pw      = new DAT[nx*ny];
    Sw      = new DAT[nx*ny];
    qx      = new DAT[(nx+1)*ny];
    qy      = new DAT[nx*(ny+1)];
    rsd_h   = new DAT[nx*ny];
    rsd_m_x = new DAT[nx*ny];
    rsd_m_y = new DAT[nx*ny];

    Tyy     = new DAT[nx*ny];
    Txx     = new DAT[nx*ny];
    Ux      = new DAT[(nx+1)*ny];
    Uy      = new DAT[nx*(ny+1)];
    Vx      = new DAT[(nx+1)*ny];
    Vy      = new DAT[nx*(ny+1)];

    std::fill_n(Ux, (nx+1)*ny, 0.0);
    std::fill_n(Uy, nx*(ny+1), 0.0);

    SetIC_GPU();
    hipDeviceSynchronize();
    SaveVTK_GPU(respath + "/sol0.vtk");

    for(int it = 1; it <= nt; it++){
        printf("\n\n =======  TIME = %lf s =======\n", it*dt);
        if(do_mech)
            M_Substep_GPU();
        if(do_flow)
            H_Substep_GPU();
        string name = respath + "/sol" + to_string(it) + ".vtk";
        SaveVTK_GPU(name);
        SaveDAT_GPU(it);
    }

    hipFree(dev_Pw);
    hipFree(dev_Sw);
    hipFree(dev_Pw_old);
    hipFree(dev_Sw_old);
    hipFree(dev_qx);
    hipFree(dev_qy);
    hipFree(dev_Krx);
    hipFree(dev_Kry);
    hipFree(dev_rsd_h);

    hipFree(dev_Txx);
    hipFree(dev_Tyy);
    hipFree(dev_Txy);
    hipFree(dev_Ux);
    hipFree(dev_Uy);
    hipFree(dev_Vx);
    hipFree(dev_Vy);
    hipFree(dev_rsd_m_x);
    hipFree(dev_rsd_m_y);

    delete [] Pw;
    delete [] Sw;
    delete [] qx;
    delete [] qy;
    delete [] rsd_h;

    delete [] Tyy;
    delete [] Txx;
    delete [] Ux;
    delete [] Uy;
}


__global__ void kernel_SetIC(DAT *Txx, DAT *Tyy, DAT *Txy,
                             DAT *Vx, DAT *Vy, DAT *Ux, DAT *Uy,
                             DAT *rsd_m_x, DAT *rsd_m_y,
                             DAT *Pw, DAT *Sw, DAT *qx, DAT *qy, DAT *Krx, DAT *Kry, DAT *rsd_h,
                             const int nx, const int ny, const DAT Lx, const DAT Ly)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;


    const DAT dx = Lx/nx, dy = dx;

    DAT x = (i+0.5)*dx, y = (j+0.5)*dy;
    // Cell variables
    if(i >= 0 && i < nx && j >= 0 && j < ny){
        //if(i*i + j*j < 400)
//        if(sqrt((Lx/2.0-x)*(Lx/2.0-x) + (Ly/2.0-y)*(Ly/2.0-y)) < 0.001)
//            Pw[i+j*nx] = 1e3;
//        else
//            Pw[i+j*nx] = -1e5;
        //DAT rad = (DAT)(i*i + j*j);
        //Pw[i+j*nx] = sqrt(rad);
        Pw[i+j*nx] = -1e5;
        rsd_h[i+j*nx] = 0.0;
        Txx[i+j*nx] = 0.0;
        Tyy[i+j*nx] = 0.0;

        rsd_m_x[i+j*nx] = 0.0;
        rsd_m_y[i+j*nx] = 0.0;
    }
    // Vertical face variables - x-fluxes, for example
    if(i >=0 && i <= nx && j >=0 && j < ny){
        int ind = i+j*(nx+1);
        qx[ind] = 0.0;
        Krx[ind] = 1.0;
        Ux[ind] = 0.0;
        Vx[ind] = 0.0;
    }
    // Horizontal face variables - y-fluxes, for example
    if(i >=0 && i < nx && j >=0 && j <= ny){
        int ind = i+j*nx;
        qy[ind] = 0.0;
        Kry[ind] = 1.0;
        Vy[ind] = 0.0;
        Uy[ind] = 0.0;
    }

    if(i >= 0 && i <= nx && j >= 0 && j <= ny){
        Txy[i+j*(nx+1)] = 0.0;
    }
}

__global__ void kernel_Update_V(DAT *Vx, DAT *Vy, DAT *Txx, DAT *Tyy, DAT *Txy, DAT *Pw, DAT *Sw,
                                const int nx, const int ny, const DAT dx, const DAT dy,
                                const DAT rho_s, const DAT g)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if(i > 0 && i < nx && j >= 0 && j <= ny-1){ // Internal faces
        DAT dTxxdx  = (Txx[i+j*nx] - Txx[i-1+j*nx]) / dx;
        DAT dPwSwdx = (Sw[i+j*nx]*Pw[i+j*nx] - Sw[i-1+j*nx]*Pw[i-1+j*nx]) / dx;

        Vx[i+j*(nx+1)]     += dt_m * (1./rho_s*(dTxxdx - 0*dPwSwdx) - g);

        if(j > 0 && j < ny-1)
            Vx[i+j*(nx+1)] += dt_m/rho_s * (Txy[i+(j+1)*(nx+1)] - Txy[i+j*(nx+1)]) / dy;
    }

    if(i >= 0 && i <= nx-1 && j > 0 && j < ny){ // Internal faces
        DAT dTyydy = (Tyy[i+j*nx] - Tyy[i+(j-1)*nx]) / dy;
        DAT dPwSwdy = (Sw[i+j*nx]*Pw[i+j*nx] - Sw[i+(j-1)*nx]*Pw[i+(j-1)*nx]) / dy;

        Vy[i+j*nx]     += dt_m * (1./rho_s*(dTyydy - 0*dPwSwdy) - g);

        if(i > 0 && i < nx-1)
            Vy[i+j*nx] += dt_m/rho_s * (Txy[i+1+j*(nx+1)] - Txy[i+j*(nx+1)]) / dx;
    }

    // BC
    if(i == 0 && j >= 0 && j <= ny-1){ // Left BCs: zero stress
        Vx[i+j*(nx+1)] += dt_m * (1./rho_s*(Txx[i+j*nx]-0.)/dx - g);
    }
//    if(i == nx && j >= 0 && j < ny){ // Right BCs: zero stress
//        Vx[i+j*(nx+1)] += dt_m/rho_s * (0.-Txx[nx-1+j*nx])/dx;
//    }
//    if(j == 0 && i >= 0 && i < nx){ // Lower BCs: stress equal to water pressure?
//        Vy[i+j*nx] += dt_m * (1./rho_s*(Tyy[i+0*nx]-Pw[i+0*nx])/dy - g);
//    }
}

__global__ void kernel_Update_U(DAT *Ux, DAT *Uy, DAT *Vx, DAT *Vy,
                                const int nx, const int ny)

{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if(i >= 0 && i <= nx && j >= 0 && j <= ny-1){
        Vx[i+j*(nx+1)] /= (1. + 1./nx);
        Ux[i+j*(nx+1)] += dt_m * Vx[i+j*(nx+1)];
    }

    if(i >= 0 && i <= nx-1 && j >= 0 && j <= ny){
        Vy[i+j*nx] /= (1. + 1./ny);
        Uy[i+j*nx] += dt_m * Vy[i+j*nx];
    }
}

__global__ void kernel_Update_Stress(DAT *Txx, DAT *Tyy, DAT *Txy, DAT *Vx, DAT *Vy,
                                     DAT *Pw, DAT *Sw, DAT *Sw_old,
                                     DAT *rsd_m_x, DAT *rsd_m_y,
                                     const int nx, const int ny,
                                     const DAT dx, const DAT dy, const DAT dt,
                                     const DAT rho_s, const DAT g, const DAT mu, const DAT lam)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;


    // Update Txy first
    if(i > 0 && i < nx && j > 0 && j < ny){
        DAT dVxdy = (Vx[i+j*(nx+1)] - Vx[i+(j-1)*(nx+1)])/dy;
        DAT dVydx = (Vy[i+j*nx]   - Vy[i-1+j*nx])/dx;
        Txy[i+j*(nx+1)] += dt_m * mu*(dVxdy + dVydx);
    }

    if(i >= 0 && i < nx && j >= 0 && j < ny){
        int ind = i+nx*j;

        DAT dVxdx = (Vx[i+1+j*(nx+1)] - Vx[i+j*(nx+1)])/dx;
        DAT dVydy = (Vy[i+(j+1)*nx]   - Vy[i+j*nx])/dy;
        DAT dSwdt = (Sw[ind] - Sw_old[ind])/dt;

        Txx[ind] += dt_m * ((2*mu+lam)*dVxdx + lam*dVydy - 0*dSwdt);
        Tyy[ind] += dt_m * ((2*mu+lam)*dVydy + lam*dVxdx - 0*dSwdt);

        if(i < nx-1 && j < ny-1){
           rsd_m_x[ind] = (Txx[i+1+j*nx]-Txx[ind])/dx
                        - 0*(Sw[i+1+j*nx]*Pw[i+1+j*nx]-Sw[ind]*Pw[ind])/dx
                        + (Txy[i+(j+1)*(nx+1)]-Txy[i+j*(nx+1)])/dy
                        - rho_s*g;

           rsd_m_y[ind] = (Tyy[i+(j+1)*nx]-Tyy[ind])/dy
                        - 0*(Sw[i+(j+1)*nx]*Pw[i+(j+1)*nx]-Sw[ind]*Pw[ind])/dy
                        + (Txy[i+1+j*(nx+1)]-Txy[i+j*(nx+1)])/dy
                        - rho_s*g;
        }
    }
}


__global__ void kernel_Compute_Sw(DAT *Pw, DAT *Sw,
                             const int nx, const int ny,
                             const DAT rhow, const DAT g,
                             const DAT vg_a, const DAT vg_m, const DAT vg_n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if(i >= 0 && i < nx && j >= 0 && j < ny){
        if(Pw[i+j*nx] >= 0.0)
            Sw[i+j*nx] = 1.0;
        else{
            Sw[i+j*nx] = pow(1.0 + pow(-vg_a/rhow/g*Pw[i+j*nx], vg_n), -vg_m);
        }
    }
}

__global__ void kernel_Compute_Q(DAT *qx, DAT *qy, DAT *Pw, DAT *Krx, DAT *Kry,
                                 const int nx, const int ny, const DAT dx, const DAT dy,
                                 const DAT K,  const DAT rhow, const DAT muw, const DAT g)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    // qx: (nx+1)xny
    // in matlab: qx(2:end-1,:)
    // 2:nx,:
    // 1:nx-1,:
    if(i > 0 && i < nx && j >= 0 && j <= ny-1){ // Internal fluxes
        qx[i+j*(nx+1)] = -rhow*K/muw*Krx[i+j*(nx+1)]*((Pw[i+j*nx] - Pw[i-1+j*nx])/dx);
        //if(i==1 && j==0)
        //    printf("qx at cell 0 = %lf\n",qx[i+j*(nx+1)]);
    }

    if(i >= 0 && i <= nx-1 && j > 0 && j < ny){ // Internal fluxes
        qy[i+j*nx] = -rhow*K/muw*Kry[i+j*nx]*((Pw[i+j*nx] - Pw[i+(j-1)*nx])/dy + rhow*g);
    }

    // Bc at lower side
    if(j == 0){
        DAT Lx = nx*dx, Ly = ny*dy;
        DAT x  =  i*dx,  y =  j*dy;
        if(x > Lx/2.-Lx/8. && x < Lx/2.+Lx/8.)
        //if(i >= 14 && i <= nx-15)
            qy[i+0*nx] = -rhow*K/muw*((Pw[i+0*nx] - 1e3)/dy + rhow*g);
        else
            qy[i+0*nx] = 0.0;
    }
    //if(j == ny)
    //    qy[i+j*nx] = 0.0;
    //if(i == 0 || i == nx)
    //    qx[i+j*(nx+1)] = 0.0;
}

__global__ void kernel_Compute_Kr(DAT *qx, DAT *qy, DAT *Pw, DAT *Sw, DAT *Krx, DAT *Kry,
                                 const int nx, const int ny, const DAT vg_m)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if(i > 0 && i < nx && j >= 0 && j <= ny-1){ // Internal faces
        DAT Swupw;
        if(Pw[i-1+j*nx] > Pw[i+j*nx])
            Swupw = Sw[i-1+j*nx];
        else
            Swupw = Sw[i+j*nx];

        // !!!! CENTRAL
        Swupw = 0.5*(Sw[i+j*nx]+Sw[i-1+j*nx]);

        Krx[i+j*(nx+1)] = sqrt(Swupw) * pow(pow(1.-pow(Swupw,1./vg_m),vg_m)-1.,2.);
    }

    if(i >= 0 && i <= nx-1 && j > 0 && j < ny){ // Internal faces
        DAT Swupw;
        if(Pw[i+(j-1)*nx] > Pw[i+j*nx]) // Todo: upwind based on head rather than pressure
            Swupw = Sw[i+(j-1)*nx];
        else
            Swupw = Sw[i+j*nx];
        if(isinf(Swupw) || isnan(Swupw)){
            printf("Bad Sw\n");
        }

        // !!!! CENTRAL
        Swupw = 0.5*(Sw[i+j*nx]+Sw[i+(j-1)*nx]);

        Kry[i+j*nx] = sqrt(Swupw) * pow(pow(1.-pow(Swupw,1./vg_m),vg_m)-1.,2.);
        if(isinf(Kry[i+j*nx]) || isnan(Kry[i+j*nx])){
            printf("Bad Kr\n");
        }
    }
}

__global__ void kernel_Update_Pw(DAT *rsd, DAT *Pw, DAT *Sw, DAT *Pw_old, DAT *Sw_old,
                                 DAT *qx, DAT *qy, const int nx, const int ny,
                                 const DAT dx,  const DAT dy,   const DAT dt,
                                 const DAT phi, const DAT rhow, const DAT sstor)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    //const DAT dt_h = 1e-2;

    if(i >= 0 && i < nx && j >= 0 && j < ny){
        int ind = i+nx*j;
        rsd[ind] = phi*rhow * (Sw[ind] - Sw_old[ind])/dt
                 + rhow*sstor * Sw[ind] * (Pw[ind] - Pw_old[ind])/dt
                 + (qx[i+1+j*(nx+1)] - qx[i+j*(nx+1)])/dx
                 + (qy[i+(j+1)*nx] - qy[i+j*nx])/dy;

        Pw[ind]  -= rsd[ind] * dt_h;
        //if(i==nx-1 && j==ny-2 && fabs(rsd[ind])>1e-18)
        //    printf("rsd = %lf\n", rsd[ind]);
        //    Pw[ind] = 1e11;
        //Pw[ind] = 1e11;
    }
}

void Problem::SaveVTK_GPU(std::string path)
{
    // Copy data from device and perform standard SaveVTK

    hipMemcpy(Pw, dev_Pw, sizeof(DAT) * nx*ny, hipMemcpyDeviceToHost);
    hipMemcpy(Sw, dev_Sw, sizeof(DAT) * nx*ny, hipMemcpyDeviceToHost);
    hipMemcpy(qx, dev_qx, sizeof(DAT) * (nx+1)*ny, hipMemcpyDeviceToHost);
    hipMemcpy(qy, dev_qy, sizeof(DAT) * nx*(ny+1), hipMemcpyDeviceToHost);

    hipMemcpy(Tyy, dev_Tyy, sizeof(DAT) * nx*ny, hipMemcpyDeviceToHost);
    hipMemcpy(Txx, dev_Txx, sizeof(DAT) * nx*ny, hipMemcpyDeviceToHost);
    hipMemcpy(Ux, dev_Ux, sizeof(DAT) * (nx+1)*ny, hipMemcpyDeviceToHost);
    hipMemcpy(Uy, dev_Uy, sizeof(DAT) * nx*(ny+1), hipMemcpyDeviceToHost);
    hipMemcpy(Vx, dev_Vx, sizeof(DAT) * (nx+1)*ny, hipMemcpyDeviceToHost);
    hipMemcpy(Vy, dev_Vy, sizeof(DAT) * nx*(ny+1), hipMemcpyDeviceToHost);

    SaveVTK(path);
}

void Problem::SaveDAT_GPU(int stepnum)
{
    hipMemcpy(Pw, dev_Pw, sizeof(DAT) * nx*ny, hipMemcpyDeviceToHost);
    hipMemcpy(Sw, dev_Sw, sizeof(DAT) * nx*ny, hipMemcpyDeviceToHost);
    hipMemcpy(qx, dev_qx, sizeof(DAT) * (nx+1)*ny, hipMemcpyDeviceToHost);
    hipMemcpy(qy, dev_qy, sizeof(DAT) * nx*(ny+1), hipMemcpyDeviceToHost);

    hipMemcpy(Tyy, dev_Tyy, sizeof(DAT) * nx*ny, hipMemcpyDeviceToHost);
    hipMemcpy(Txx, dev_Txx, sizeof(DAT) * nx*ny, hipMemcpyDeviceToHost);
    hipMemcpy(Ux, dev_Ux, sizeof(DAT) * (nx+1)*ny, hipMemcpyDeviceToHost);
    hipMemcpy(Uy, dev_Uy, sizeof(DAT) * nx*(ny+1), hipMemcpyDeviceToHost);
    hipMemcpy(Vx, dev_Vx, sizeof(DAT) * (nx+1)*ny, hipMemcpyDeviceToHost);
    hipMemcpy(Vy, dev_Vy, sizeof(DAT) * nx*(ny+1), hipMemcpyDeviceToHost);

    std::string path = "C:\\Users\\Denis\\Documents\\msu_thmc\\MATLAB\\res_gpu\\";
    FILE *f;
    std::string fname;

    fname = path + "Ux" + std::to_string(stepnum) + ".dat";
    f = fopen(fname.c_str(), "wb");
    fwrite(Ux, sizeof(DAT), (nx+1)*ny, f);
    fclose(f);

    fname = path + "Uy" + std::to_string(stepnum) + ".dat";
    f = fopen(fname.c_str(), "wb");
    fwrite(Uy, sizeof(DAT), (ny+1)*nx, f);
    fclose(f);

    fname = path + "Sw" + std::to_string(stepnum) + ".dat";
    f = fopen(fname.c_str(), "wb");
    fwrite(Sw, sizeof(DAT), ny*nx, f);
    fclose(f);
}
