#include "hip/hip_runtime.h"
#include "header.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

using namespace std;

#define BLOCK_DIM 16
#define dt_h      5e7
#define dt_m      1e-6

void FindMax(DAT *dev_arr, DAT *max, int size);

__global__ void kernel_SetIC(DAT *Pl, DAT *Pg, DAT *Sl,
                             DAT *Kx, DAT *Ky,
                             DAT *Krlx, DAT *Krly, DAT *Krgx, DAT *Krgy,
                             DAT *qlx, DAT *qly, DAT *qgx, DAT *qgy,
                             DAT *phi,
                             DAT *rsd_l, DAT *rsd_g,
                             const DAT K0,
                             const int nx, const int ny, const DAT Lx, const DAT Ly);

__global__ void kernel_Compute_Q();

__global__ void kernel_Compute_K();

__global__ void kernel_Update_P();

__global__ void kernel_Update_Poro();

void FindMax(DAT *dev_arr, DAT *max, int size)
{
    hipblasHandle_t handle;
    hipblasStatus_t stat;
    hipblasCreate(&handle);

    int maxind = 0;
    stat = hipblasIdamax(handle, size, dev_arr, 1, &maxind);
    //stat = hipblasIsamax(handle, size, dev_arr, 1, &maxind);
    if (stat != HIPBLAS_STATUS_SUCCESS)
        printf("Max failed\n");


    hipMemcpy(max, dev_arr+maxind-1, sizeof(DAT), hipMemcpyDeviceToHost);

    hipblasDestroy(handle);
}

void Problem::SetIC_GPU()
{
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid((nx+dimBlock.x-1)/dimBlock.x, (ny+dimBlock.y-1)/dimBlock.y);
    printf("Launching %dx%d blocks of %dx%d threads\n", (nx+1+dimBlock.x-1)/dimBlock.x,
           (ny+1+dimBlock.y-1)/dimBlock.y, BLOCK_DIM, BLOCK_DIM);
    kernel_SetIC<<<dimGrid,dimBlock>>>(dev_Pl, dev_Pg, dev_Sl,
                                       dev_Kx, dev_Ky,
                                       dev_Krlx, dev_Krly, dev_Krgx, dev_Krgy,
                                       dev_qlx, dev_qly, dev_qgx, dev_qgy,
                                       dev_phi,
                                       dev_rsd_l, dev_rsd_l,
                                       K0,
                                       nx, ny, Lx, Ly);
    hipError_t err = hipGetLastError();
    if(err != 0)
        printf("Error %x at SetIC\n", err);
}


void Problem::Compute_Q_GPU()
{
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid((nx+1+dimBlock.x-1)/dimBlock.x, (ny+1+dimBlock.y-1)/dimBlock.y);
    kernel_Compute_Q<<<dimGrid,dimBlock>>>();
    hipError_t err = hipGetLastError();
    if(err != 0)
        printf("Error %x at Q\n", err);
}

void Problem::Compute_K_GPU()
{
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid((nx+1+dimBlock.x-1)/dimBlock.x, (ny+1+dimBlock.y-1)/dimBlock.y);
    kernel_Compute_K<<<dimGrid,dimBlock>>>();
    hipError_t err = hipGetLastError();
    if(err != 0)
        printf("Error %x at K\n", err);
}

void Problem::Compute_Kr_GPU()
{
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid((nx+1+dimBlock.x-1)/dimBlock.x, (ny+1+dimBlock.y-1)/dimBlock.y);
}

void Problem::Compute_S_GPU()
{
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid((nx+1+dimBlock.x-1)/dimBlock.x, (ny+1+dimBlock.y-1)/dimBlock.y);
}

void Problem::Update_P_GPU()
{
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid((nx+dimBlock.x-1)/dimBlock.x, (ny+dimBlock.y-1)/dimBlock.y);
    kernel_Update_P<<<dimGrid,dimBlock>>>();
    hipError_t err = hipGetLastError();
    if(err != 0)
        printf("Error %x at Pf\n", err);
}

void Problem::Update_Poro()
{
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid((nx+dimBlock.x-1)/dimBlock.x, (ny+dimBlock.y-1)/dimBlock.y);
    kernel_Update_Poro<<<dimGrid,dimBlock>>>();
    hipError_t err = hipGetLastError();
    if(err != 0)
        printf("Error %x at Poro\n", err);
}

void Problem::H_Substep_GPU()
{
    printf("Flow\n");
    fflush(stdout);
    DAT err_l = 1, err_g = 1, err_l_old, err_g_old;
    for(int nit = 1; nit <= niter; nit++){
        Compute_K_GPU();
        Compute_S_GPU();
        Compute_Kr_GPU();
        Compute_Q_GPU();
        Update_P_GPU();
        if(nit%10000 == 0 || nit == 1){
            err_l_old = err_l;
            err_g_old = err_g;
            FindMax(dev_rsd_l, &err_l, nx*ny);
            FindMax(dev_rsd_g, &err_g, nx*ny);
            printf("iter %d: r_l = %e, r_g = %e\n", nit, err_l, err_g);
            fflush(stdout);
            if((    (err_l<eps_a_h && err_g<eps_a_h) ||
                    (fabs(err_l-err_l_old) < 1e-15 && fabs(err_g-err_g_old) < 1e-15))
                && nit > 10000){
                printf("Flow converged in %d it.: r_l = %e, r_g = %e\n", nit, err_l, err_g);
                break;
            }
        }
    }
    //Update_Poro();
}

void Problem::SolveOnGPU()
{
    hipEvent_t tbeg, tend;
    hipEventCreate(&tbeg);
    hipEventCreate(&tend);
    hipEventRecord(tbeg);
    hipMalloc((void**)&dev_Pl,     sizeof(DAT) * nx*ny);
    hipMalloc((void**)&dev_Pl_old, sizeof(DAT) * nx*ny);
    hipMalloc((void**)&dev_Pg,     sizeof(DAT) * nx*ny);
    hipMalloc((void**)&dev_Pg_old, sizeof(DAT) * nx*ny);
    hipMalloc((void**)&dev_Sl,     sizeof(DAT) * nx*ny);
    hipMalloc((void**)&dev_Sl_old, sizeof(DAT) * nx*ny);
    hipMalloc((void**)&dev_qlx,    sizeof(DAT) * (nx+1)*ny);
    hipMalloc((void**)&dev_qly,    sizeof(DAT) * nx*(ny+1));
    hipMalloc((void**)&dev_qgx,    sizeof(DAT) * (nx+1)*ny);
    hipMalloc((void**)&dev_qgy,    sizeof(DAT) * nx*(ny+1));
    hipMalloc((void**)&dev_Kx,     sizeof(DAT) * (nx+1)*ny);
    hipMalloc((void**)&dev_Ky,     sizeof(DAT) * nx*(ny+1));
    hipMalloc((void**)&dev_Krlx,   sizeof(DAT) * (nx+1)*ny);
    hipMalloc((void**)&dev_Krly,   sizeof(DAT) * nx*(ny+1));
    hipMalloc((void**)&dev_Krgx,   sizeof(DAT) * (nx+1)*ny);
    hipMalloc((void**)&dev_Krgy,   sizeof(DAT) * nx*(ny+1));
    hipMalloc((void**)&dev_phi,    sizeof(DAT) * nx*ny);
    hipMalloc((void**)&dev_rsd_l,  sizeof(DAT) * nx*ny);
    hipMalloc((void**)&dev_rsd_g,  sizeof(DAT) * nx*ny);
    hipEventRecord(tbeg);

    printf("Allocated on GPU\n");

    // Still needed for VTK saving
    Pl      = new DAT[nx*ny];
    qlx      = new DAT[(nx+1)*ny];
    qly      = new DAT[nx*(ny+1)];
    Kx      = new DAT[(nx+1)*ny];
    Ky      = new DAT[nx*(ny+1)];
    phi     = new DAT[nx*ny];

    SetIC_GPU();
    hipDeviceSynchronize();

    SaveVTK_GPU(respath + "/sol0.vtk");

    for(int it = 1; it <= nt; it++){
        printf("\n\n =======  TIME STEP %d, T = %lf s =======\n", it, it*dt);
        if(do_mech)
;//            M_Substep_GPU();
        hipMemcpy(dev_Pl_old, dev_Pl, sizeof(DAT) * nx*ny, hipMemcpyDeviceToDevice);
        hipMemcpy(dev_Pg_old, dev_Pg, sizeof(DAT) * nx*ny, hipMemcpyDeviceToDevice);
        hipMemcpy(dev_Sl_old, dev_Sl, sizeof(DAT) * nx*ny, hipMemcpyDeviceToDevice);
        H_Substep_GPU();
        string name = respath + "/sol" + to_string(it) + ".vtk";
        SaveVTK_GPU(name);
        SaveDAT_GPU(it);
    }

    hipFree(dev_Pl);
    hipFree(dev_Pl_old);
    hipFree(dev_Pg);
    hipFree(dev_Pg_old);
    hipFree(dev_Sl);
    hipFree(dev_Sl_old);
    hipFree(dev_qlx);
    hipFree(dev_qly);
    hipFree(dev_qgx);
    hipFree(dev_qgy);
    hipFree(dev_Kx);
    hipFree(dev_Ky);
    hipFree(dev_Krlx);
    hipFree(dev_Krly);
    hipFree(dev_Krgx);
    hipFree(dev_Krgy);
    hipFree(dev_phi);
    hipFree(dev_rsd_l);
    hipFree(dev_rsd_g);

    hipEventRecord(tend);
    hipEventSynchronize(tend);

    float comptime = 0.0;
    hipEventElapsedTime(&comptime, tbeg, tend);
    printf("\nComputation time = %f s\n", comptime/1e3);

    delete [] Pl;
    delete [] qlx;
    delete [] qly;
    delete [] Kx;
    delete [] Ky;
    delete [] phi;
}


__global__ void kernel_SetIC(DAT *Pl, DAT *Pg, DAT *Sl,
                             DAT *Kx, DAT *Ky,
                             DAT *Krlx, DAT *Krly, DAT *Krgx, DAT *Krgy,
                             DAT *qlx, DAT *qly, DAT *qgx, DAT *qgy,
                             DAT *phi,
                             DAT *rsd_l, DAT *rsd_g,
                             const DAT K0,
                             const int nx, const int ny,
                             const DAT Lx, const DAT Ly
                             )
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;


    const DAT dx = Lx/nx, dy = Ly/ny;

    DAT x = (i+0.5)*dx, y = (j+0.5)*dy;
    // Cell variables
    if(i >= 0 && i < nx && j >= 0 && j < ny){
        if(sqrt((Lx/2.0-x)*(Lx/2.0-x) + (Ly/2.0-y)*(Ly/2.0-y)) < 0.001)
            Pl[i+j*nx] = 10e6;
        else
            Pl[i+j*nx] = 8e6;

        Pg[i+j*nx] = 8e6;
        phi[i+j*nx] = 0.16;
        rsd_l[i+j*nx] = 0.0;
        rsd_g[i+j*nx] = 0.0;
    }
    // Vertical face variables - x-fluxes, for example
    if(i >= 0 && i <= nx && j >= 0 && j < ny){
        int ind = i+j*(nx+1);
        qlx[ind] = 0.0;
        qgx[ind] = 0.0;
        Kx[ind] = K0;
        Krlx[ind] = 1.0;
        Krgx[ind] = 1.0;
    }
    // Horizontal face variables - y-fluxes, for example
    if(i >= 0 && i < nx && j >= 0 && j <= ny){
        int ind = i+j*nx;
        qly[ind] = 0.0;
        qgy[ind] = 0.0;
        Ky[ind] = K0;
        Krly[ind] = 1.0;
        Krgy[ind] = 1.0;
    }
}



__global__ void kernel_Compute_Q()
{

}

__global__ void kernel_Compute_K()
{

}

__global__ void kernel_Update_P()
{

}

__global__ void kernel_Update_Poro()
{

}

void Problem::SaveVTK_GPU(std::string path)
{
    // Copy data from device and perform standard SaveVTK

    hipMemcpy(Pl,  dev_Pl, sizeof(DAT) * nx*ny, hipMemcpyDeviceToHost);
    hipMemcpy(qlx, dev_qlx, sizeof(DAT) * (nx+1)*ny, hipMemcpyDeviceToHost);
    hipMemcpy(qly, dev_qly, sizeof(DAT) * nx*(ny+1), hipMemcpyDeviceToHost);
    hipMemcpy(Kx,  dev_Kx, sizeof(DAT) * (nx+1)*ny, hipMemcpyDeviceToHost);
    hipMemcpy(Ky,  dev_Ky, sizeof(DAT) * nx*(ny+1), hipMemcpyDeviceToHost);
    hipMemcpy(phi, dev_phi, sizeof(DAT) * nx*ny, hipMemcpyDeviceToHost);

    SaveVTK(path);
}

void Problem::SaveDAT_GPU(int stepnum)
{
    hipMemcpy(Pl, dev_Pl, sizeof(DAT) * nx*ny, hipMemcpyDeviceToHost);
    hipMemcpy(Kx, dev_Kx, sizeof(DAT) * (nx+1)*ny, hipMemcpyDeviceToHost);
    hipMemcpy(Ky, dev_Ky, sizeof(DAT) * nx*(ny+1), hipMemcpyDeviceToHost);

    std::string path = "C:\\Users\\Denis\\Documents\\msu_thmc\\MATLAB\\res_gpu\\shale_1phase\\";
    FILE *f;
    std::string fname;

    fname = path + "Pl" + std::to_string(stepnum) + ".dat";
    f = fopen(fname.c_str(), "wb");
    fwrite(Pl, sizeof(DAT), ny*nx, f);
    fclose(f);

    fname = path + "Ky" + std::to_string(stepnum) + ".dat";
    f = fopen(fname.c_str(), "wb");
    fwrite(Ky, sizeof(DAT), (ny+1)*nx, f);
    fclose(f);

    fname = path + "Kx" + std::to_string(stepnum) + ".dat";
    f = fopen(fname.c_str(), "wb");
    fwrite(Kx, sizeof(DAT), (nx+1)*ny, f);
    fclose(f);
}
